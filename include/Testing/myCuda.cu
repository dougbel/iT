#include "hip/hip_runtime.h"
  #include <thrust/reduce.h>
  #include <thrust/host_vector.h>
  #include <thrust/device_vector.h>

__constant__ double PI = 3.141592653589;

    //  nn              =>    d_nnData        ==> array of nearest point in scene from a given  provenance vector
    //  kp              =>    kpData          ==> key point to tests, 512 * 8 (samples calculated and rotated)
    //  pv              =>    pvData          ==> provenance vectors repited as many as orientations
    //  tp              =>    d_testPoint     ==> testPoint
    //  C               =>    d_C             ==> OUTPUT this will be the result container 
    //  start           =>    start           ==> first number of sample point to test in the batch
    //  end             =>    end             ==> las number of sample point  to test in the batch
    //  weights         =>    wData           ==> provenance vector norms mapped, the smaller vector the bigger value in map [1,0]
    //  comp            =>    4               ==> this value always is 4
    //  mags            =>    mData           ==> provenance vector norms
    //  a_th            =>    object->agg_th  ==> threshold read from parameters.txt (VectorDiff)
    //  ppCentroid      =>    ppC             ==> it only have "1" as value, possibly to detected multiple affordances in future
    //  startppCentroid =>    startppC        ==> index of position 0-4095 (asociated with ppC), possibly to detected multiple affordances in future
    //  ppCentroidData  =>    ppCData         ==> this ha 4 values: affordance id, orientation, #ofKeypoint, and a value to align
  __global__ void bayesianKernel(float *nn,float *kp, float *pv, float *tp, float *C, int start, int end, int comp, float *weights, float *mags, float a_th, int *ppCentroid, int* startppCentroid, float *ppCentroidData){

    //I think I only need row
    int inner_ele = blockIdx.y*blockDim.y+threadIdx.y;    //This goes 0-2048
    int actual_ele=inner_ele + start; //This goes 0-2048 for now, could get larger
    //Get the actual_ele neighbour and compute vectors and stuff
    float xt= nn[actual_ele*comp+0]-(tp[0]+kp[actual_ele*3+0]);
    float yt= nn[actual_ele*comp+1]-(tp[1]+kp[actual_ele*3+1]);
    float zt= nn[actual_ele*comp+2]-(tp[2]+kp[actual_ele*3+2]);
    for (int i=0;i<ppCentroid[actual_ele];i++)
    {
      int idx=startppCentroid[actual_ele]+i;  //0-2969 for now, could get larger
      int or_id=ppCentroidData[idx*comp+1];
      int pv_id=ppCentroidData[idx*comp+2];


      float angle=or_id*2*PI/8;
      float xpv=pv[idx*3+0]*cos(angle)-pv[idx*3+1]*sin(angle);
      float ypv=sin(angle)*pv[idx*3+0]+cos(angle)*pv[idx*3+1];
      float zpv=pv[idx*3+2];


      float diff=sqrt(((xt-xpv)*(xt-xpv))+((yt-ypv)*(yt-ypv))+((zt-zpv)*(zt-zpv)))/mags[idx];  //This is the difference as proportion of expected magnitude

      //Likelihood is the sample from a normal distribution with mean 0 and std=0.1/weighs;
      float sigma=a_th*(1+weights[idx]);
      
      float likelyhood=expf(-(diff*diff)/(2*sigma*sigma));
      
      C[idx]=likelyhood*weights[idx];


    }
  
  }

  void bayesian_scores(float *nn,float *kp, float *pv, float *tp, float *C, int start, int end, int comp, float *weights, float *mags, float a_th, int *ppCentroid, int* startppCentroid, float *ppCentroidData){
    int maxThreads=128;      //From tables
    int N=end-start;
    dim3 threadsPerBlock(1, maxThreads);  //1x128
    dim3 blocksPerGrid(1, N/maxThreads);  //1x(4096/128) => 1x32
    bayesianKernel<<<blocksPerGrid,threadsPerBlock>>>(nn, kp, pv, tp, C, start, end, comp, weights, mags, a_th, ppCentroid, startppCentroid, ppCentroidData);
    hipDeviceSynchronize();
  }
